#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>

static const unsigned int tileSize = 8;
static const unsigned int maxPrimitivesPerTile = 256;

namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 objectPos;
        glm::vec4 worldPos;
        glm::vec4 viewPos;
        glm::vec4 clipPos;
        glm::vec3 windowPos;
        glm::vec3 objectNor;
        glm::vec3 worldNor;
        glm::vec3 viewNor;
        glm::vec3 color;
		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own


		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		// int texWidth, texHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
        float depth;
		glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		// glm::vec3 eyePos;	// eye space position used for shading
		// glm::vec3 eyeNor;
		// VertexAttributeTexcoord texcoord0;
		// TextureData* dev_diffuseTex;
		// ...
	};

    struct Tile {
        unsigned int numPrimitives;
        unsigned int primitiveId[maxPrimitivesPerTile];
    };

    struct SceneInfo {
        unsigned int numPrimitives;
    };

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;

static int width = 0;
static int height = 0;

static SceneInfo sceneInfo;

static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static Tile *dev_tileBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;


////////////////////////////////////////////////////////////////
///                           Debug                          ///
////////////////////////////////////////////////////////////////
__global__
void debug_printPrimitives(int numPrimitives, Primitive *dev_primitives)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid<numPrimitives)
    {
        printf("---Primitive %d---\n%f,%f,%f,%f\n%f,%f,%f,%f\n%f,%f,%f,%f\n------\n\n",
               pid,
               dev_primitives[pid].v[0].windowPos.x,
               dev_primitives[pid].v[0].windowPos.y,
               dev_primitives[pid].v[0].windowPos.z,
               1.0,//dev_primitives[pid].v[0].windowPos.w,
               dev_primitives[pid].v[1].windowPos.x,
               dev_primitives[pid].v[1].windowPos.y,
               dev_primitives[pid].v[1].windowPos.z,
               1.0,//dev_primitives[pid].v[1].windowPos.w,
               dev_primitives[pid].v[2].windowPos.x,
               dev_primitives[pid].v[2].windowPos.y,
               dev_primitives[pid].v[2].windowPos.z,
               1.0//dev_primitives[pid].v[2].windowPos.w
               );
    }
}


////////////////////////////////////////////////////////////////
///                      Render Pipeline                     ///
////////////////////////////////////////////////////////////////

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = fragmentBuffer[index].color;

        // framebuffer[index] = glm::vec3(1,1,1);

        // TODO: add your fragment shader code here

    }
}

__global__
void _clearBuffer(Tile *dev_tileBuffer, Fragment *dev_fragmentBuffer, int width, int height, int tileSize)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int maxTileNumX = (width + tileSize - 1)/tileSize;
    const int maxTileNumY = (height + tileSize - 1)/tileSize;

	if (x < width && y < height)
	{
        int index = x + (y * width);
        dev_fragmentBuffer[index].depth = 1;
        dev_fragmentBuffer[index].color = glm::vec3(0,0,0);
	}

    if (x < maxTileNumX && y < maxTileNumY)
    {
        int index = x + y * maxTileNumX;
        dev_tileBuffer[index].numPrimitives=0;
    }
}


__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 M, glm::mat4 V, glm::mat4 P,
	int width, int height) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space

        VertexOut& vertexOut = primitive.dev_verticesOut[vid];
        vertexOut.objectPos = glm::vec4(primitive.dev_position[vid],1);
        vertexOut.worldPos = M * vertexOut.objectPos;
        vertexOut.viewPos = V * vertexOut.worldPos;
        vertexOut.clipPos = P * vertexOut.viewPos;
        vertexOut.clipPos /= vertexOut.clipPos.w;
        vertexOut.windowPos = glm::vec3((vertexOut.clipPos.x + 1.0f) * 0.5f * width,
                                        (1.0f - vertexOut.clipPos.y) * 0.5f * height,
                                        vertexOut.clipPos.z);
        vertexOut.objectNor = primitive.dev_normal[vid];
        vertexOut.worldNor = glm::transpose(glm::inverse(glm::mat3(M))) * vertexOut.objectNor;
        vertexOut.viewNor = glm::transpose(glm::inverse(glm::mat3(V))) * vertexOut.worldNor;




		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array
		
	}
}


__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
        int vid;
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES) {
			pid = iid / (int)primitive.primitiveType;
            vid = iid % (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[vid]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
            dev_primitives[pid + curPrimitiveBeginId].v[vid].color = glm::vec3(vid==0,vid==1,vid==2);
            dev_primitives[pid + curPrimitiveBeginId].primitiveType = primitive.primitiveType;
		}


		// TODO: other primitive types (point, line)
	}
	
}

__global__
void _generateTileBuffer(int numPrimitives, Primitive* dev_primitives, Tile* dev_tileBuffer, int width, int height, int tileSize)
{
    const unsigned int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if(pid<numPrimitives){
        const int maxTileNumX = (width + tileSize - 1)/tileSize;
        const int maxTileNumY = (height + tileSize - 1)/tileSize;
        const Primitive & primitive = dev_primitives[pid];
        const glm::vec3 (&pos)[] = {primitive.v[0].windowPos,
                                    primitive.v[1].windowPos,
                                    primitive.v[2].windowPos};

        const AABB bound = getAABBForTriangle(pos);
        const int minTileIdX = glm::max(((int)glm::round(bound.min.x))/tileSize,0);
        const int maxTileIdX = glm::min(((int)glm::round(bound.max.x))/tileSize,maxTileNumX-1);
        const int minTileIdY = glm::max(((int)glm::round(bound.min.y))/tileSize,0);
        const int maxTileIdY = glm::min(((int)glm::round(bound.max.y))/tileSize,maxTileNumY-1);

        for(int x=minTileIdX;x<=maxTileIdX;x++)
        {
            for(int y=minTileIdY;y<=maxTileIdY;y++)
            {
                const unsigned int tid = x + y * maxTileNumX;
                const unsigned int id = atomicAdd(&(dev_tileBuffer[tid].numPrimitives),1);
                if(id<maxPrimitivesPerTile)
                    dev_tileBuffer[tid].primitiveId[id] = pid;
            }
        }
    }
}

__global__
void _rasterize(Primitive* dev_primitives, Tile* dev_tileBuffer, Fragment* dev_fragmentBuffer, int width, int height, int tileSize)
{
    extern __shared__ Fragment tileFragment[];

    const int maxTileNumX = (width + tileSize - 1)/tileSize;
    const int maxTileNumY = (height + tileSize - 1)/tileSize;
    const int & tileIdX = blockIdx.x;
    const int & tileIdY = blockIdx.y;
    const int tileId = tileIdX + tileIdY * maxTileNumX;
    const int & tilePosX = threadIdx.x;
    const int & tilePosY = threadIdx.y;
    const int tilePos = tilePosX + tilePosY * tileSize;
    const int posX = tileIdX * tileSize + tilePosX;
    const int posY = tileIdY * tileSize + tilePosY;
    const int pos = posX + posY * width;

    if(tileIdX>=maxTileNumX||tileIdY>=maxTileNumY) return;
    if(posX>=width||posY>=height) return;

    tileFragment[tilePos] = dev_fragmentBuffer[pos]; // Copy from global memory;
    int maxPrimitiveIdIndex = glm::min(dev_tileBuffer[tileId].numPrimitives,maxPrimitivesPerTile);

    for(int primitiveIdIndex=0;primitiveIdIndex<maxPrimitiveIdIndex;primitiveIdIndex++)
    {
        const Primitive & primitive = dev_primitives[dev_tileBuffer[tileId].primitiveId[primitiveIdIndex]];
        const glm::vec3 (&primitivePos)[] = {primitive.v[0].windowPos,
                                             primitive.v[1].windowPos,
                                             primitive.v[2].windowPos};
        glm::vec3 baryCoords = calculateBarycentricCoordinate(primitivePos, glm::vec2(posX, posY));
        bool isInsideTriangle = isBarycentricCoordInBounds(baryCoords);
        if (isInsideTriangle)
        {
            float depth = -getZAtCoordinate(baryCoords, primitivePos);
            if(depth<tileFragment[tilePos].depth)
            {
                tileFragment[tilePos].color = glm::vec3(1,1,1);
                tileFragment[tilePos].depth = depth;
            } // No need to use atomic because no data race happen
        }
        __syncthreads(); // Ensure all threads are rasterizing the same primitive
    }

    // Copy data from shared memory to global memory
    // one thread per fragment
    dev_fragmentBuffer[pos] = tileFragment[tilePos];
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & M, const glm::mat4 & V, const glm::mat4 & P) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
        int curPrimitiveBeginId = 0; // change static to non-static
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly<<<numBlocksForVertices, numThreadsPerBlock>>>
                (p->numVertices,
                 *p,
                 M, V, P,
                 width,
                 height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly<<<numBlocksForIndices, numThreadsPerBlock>>>
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

        // debug_printPrimitives<<<curPrimitiveBeginId,1>>>(curPrimitiveBeginId,dev_primitives);
		checkCUDAError("Vertex Processing and Primitive Assembly");
	}
	
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    _clearBuffer<<<blockCount2d, blockSize2d>>>(dev_tileBuffer, dev_fragmentBuffer, width, height, tileSize);
    checkCUDAError("_clearBuffer");

    // TODO: rasterize
    {
        dim3 numThreadsPerBlock = 128;
        dim3 numBlocks = (sceneInfo.numPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x;
        _generateTileBuffer<<<numBlocks,numThreadsPerBlock>>>
                (sceneInfo.numPrimitives,
                 dev_primitives,
                 dev_tileBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("_generateTileBuffer");

        numThreadsPerBlock = dim3(tileSize, tileSize);
        numBlocks = dim3((width + tileSize - 1)/tileSize,(height + tileSize - 1)/tileSize);
        _rasterize<<<numBlocks,numThreadsPerBlock,tileSize*tileSize*sizeof(Fragment)>>>
                (dev_primitives,
                 dev_tileBuffer,
                 dev_fragmentBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("_rasterize");

    }



    // Copy depthbuffer colors into framebuffer
	render<<<blockCount2d, blockSize2d>>>(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}




////////////////////////////////////////////////////////////////
/// Functions that only be called when program start or exit ///
////////////////////////////////////////////////////////////////


/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_fragmentBuffer);
    hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
    hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_tileBuffer);
    hipMalloc(&dev_tileBuffer, ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
    hipMemset(dev_tileBuffer, 0, ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

    auto it(mesh2PrimitivesMap.begin());
    auto itEnd(mesh2PrimitivesMap.end());
    for (; it != itEnd; ++it) {
        for (auto p = it->second.begin(); p != it->second.end(); ++p) {
            hipFree(p->dev_indices);
            hipFree(p->dev_position);
            hipFree(p->dev_normal);
            hipFree(p->dev_texcoord0);
            hipFree(p->dev_diffuseTex);

            hipFree(p->dev_verticesOut);


            //TODO: release other attributes and materials
        }
    }

    ////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_fragmentBuffer);
    dev_fragmentBuffer = NULL;

    hipFree(dev_tileBuffer);
    dev_tileBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterize Free");
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {

    // Attribute (vec3 position)
    // component (3 * float)
    // byte (4 * byte)

    // id of component
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i < N) {
        int count = i / n;
        int offset = i - count * n;	// which component of the attribute

        for (int j = 0; j < componentTypeByteSize; j++) {

            dev_dst[count * componentTypeByteSize * n
                    + offset * componentTypeByteSize
                    + j]

                    =

                    dev_src[byteOffset
                            + count * (byteStride == 0 ? componentTypeByteSize * n : byteStride)
                            + offset * componentTypeByteSize
                            + j];
        }
    }
}

__global__
void _nodeMatrixTransform(
        int numVertices,
        VertexAttributePosition* position,
        VertexAttributeNormal* normal,
        glm::mat4 MV, glm::mat3 MV_normal) {

    // vertex id
    int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (vid < numVertices) {
        position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
        normal[vid] = glm::normalize(MV_normal * normal[vid]);
    }
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {

    glm::mat4 curMatrix(1.0);

    const std::vector<double> &m = n.matrix;
    if (m.size() > 0) {
        // matrix, copy it

        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                curMatrix[i][j] = (float)m.at(4 * i + j);
            }
        }
    } else {
        // no matrix, use rotation, scale, translation

        if (n.translation.size() > 0) {
            curMatrix[3][0] = n.translation[0];
            curMatrix[3][1] = n.translation[1];
            curMatrix[3][2] = n.translation[2];
        }

        if (n.rotation.size() > 0) {
            glm::mat4 R;
            glm::quat q;
            q[0] = n.rotation[0];
            q[1] = n.rotation[1];
            q[2] = n.rotation[2];

            R = glm::mat4_cast(q);
            curMatrix = curMatrix * R;
        }

        if (n.scale.size() > 0) {
            curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
        }
    }

    return curMatrix;
}

void traverseNode (
        std::map<std::string, glm::mat4> & n2m,
        const tinygltf::Scene & scene,
        const std::string & nodeString,
        const glm::mat4 & parentMatrix
)
{
    const tinygltf::Node & n = scene.nodes.at(nodeString);
    glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
    n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

    auto it = n.children.begin();
    auto itEnd = n.children.end();

    for (; it != itEnd; ++it) {
        traverseNode(n2m, scene, *it, M);
    }
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

    int totalNumPrimitives = 0; // change static to non-static

    std::map<std::string, BufferByte*> bufferViewDevPointers;

    // 1. copy all `bufferViews` to device memory
    {
        std::map<std::string, tinygltf::BufferView>::const_iterator it(
                scene.bufferViews.begin());
        std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
                scene.bufferViews.end());

        for (; it != itEnd; it++) {
            const std::string key = it->first;
            const tinygltf::BufferView &bufferView = it->second;
            if (bufferView.target == 0) {
                continue; // Unsupported bufferView.
            }

            const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

            BufferByte* dev_bufferView;
            hipMalloc(&dev_bufferView, bufferView.byteLength);
            hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

            checkCUDAError("Set BufferView Device Mem");

            bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

        }
    }



    // 2. for each mesh:
    //		for each primitive:
    //			build device buffer of indices, materail, and each attributes
    //			and store these pointers in a map
    {
        std::map<std::string, glm::mat4> nodeString2Matrix;
        auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

        {
            auto it = rootNodeNamesList.begin();
            auto itEnd = rootNodeNamesList.end();
            for (; it != itEnd; ++it) {
                traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
            }
        }


        // parse through node to access mesh

        auto itNode = nodeString2Matrix.begin();
        auto itEndNode = nodeString2Matrix.end();
        for (; itNode != itEndNode; ++itNode) {

            const tinygltf::Node & N = scene.nodes.at(itNode->first);
            const glm::mat4 & matrix = itNode->second;
            const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

            auto itMeshName = N.meshes.begin();
            auto itEndMeshName = N.meshes.end();

            for (; itMeshName != itEndMeshName; ++itMeshName) {

                const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

                auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
                std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

                // for each primitive
                for (size_t i = 0; i < mesh.primitives.size(); i++) {
                    const tinygltf::Primitive &primitive = mesh.primitives[i];

                    if (primitive.indices.empty())
                        return;

                    // TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
                    VertexIndex* dev_indices = NULL;
                    VertexAttributePosition* dev_position = NULL;
                    VertexAttributeNormal* dev_normal = NULL;
                    VertexAttributeTexcoord* dev_texcoord0 = NULL;

                    // ----------Indices-------------

                    const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
                    const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
                    BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

                    // assume type is SCALAR for indices
                    int n = 1;
                    int numIndices = indexAccessor.count;
                    int componentTypeByteSize = sizeof(VertexIndex);
                    int byteLength = numIndices * n * componentTypeByteSize;

                    dim3 numThreadsPerBlock(128);
                    dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    hipMalloc(&dev_indices, byteLength);
                    _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                            numIndices,
                            (BufferByte*)dev_indices,
                            dev_bufferView,
                            n,
                            indexAccessor.byteStride,
                            indexAccessor.byteOffset,
                            componentTypeByteSize);


                    checkCUDAError("Set Index Buffer");


                    // ---------Primitive Info-------

                    // Warning: LINE_STRIP is not supported in tinygltfloader
                    int numPrimitives;
                    PrimitiveType primitiveType;
                    switch (primitive.mode) {
                        case TINYGLTF_MODE_TRIANGLES:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices / 3;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_STRIP:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_FAN:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_LINE:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices / 2;
                            break;
                        case TINYGLTF_MODE_LINE_LOOP:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices + 1;
                            break;
                        case TINYGLTF_MODE_POINTS:
                            primitiveType = PrimitiveType::Point;
                            numPrimitives = numIndices;
                            break;
                        default:
                            // output error
                            break;
                    };


                    // ----------Attributes-------------

                    auto it(primitive.attributes.begin());
                    auto itEnd(primitive.attributes.end());

                    int numVertices = 0;
                    // for each attribute
                    for (; it != itEnd; it++) {
                        const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
                        const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

                        int n = 1;
                        if (accessor.type == TINYGLTF_TYPE_SCALAR) {
                            n = 1;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC2) {
                            n = 2;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC3) {
                            n = 3;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC4) {
                            n = 4;
                        }

                        BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
                        BufferByte ** dev_attribute = NULL;

                        numVertices = accessor.count;
                        int componentTypeByteSize;

                        // Note: since the type of our attribute array (dev_position) is static (float32)
                        // We assume the glTF model attribute type are 5126(FLOAT) here

                        if (it->first.compare("POSITION") == 0) {
                            componentTypeByteSize = sizeof(VertexAttributePosition) / n;
                            dev_attribute = (BufferByte**)&dev_position;
                        }
                        else if (it->first.compare("NORMAL") == 0) {
                            componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
                            dev_attribute = (BufferByte**)&dev_normal;
                        }
                        else if (it->first.compare("TEXCOORD_0") == 0) {
                            componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
                            dev_attribute = (BufferByte**)&dev_texcoord0;
                        }

                        std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

                        dim3 numThreadsPerBlock(128);
                        dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                        int byteLength = numVertices * n * componentTypeByteSize;
                        hipMalloc(dev_attribute, byteLength);

                        _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                                n * numVertices,
                                *dev_attribute,
                                dev_bufferView,
                                n,
                                accessor.byteStride,
                                accessor.byteOffset,
                                componentTypeByteSize);

                        std::string msg = "Set Attribute Buffer: " + it->first;
                        checkCUDAError(msg.c_str());
                    }

                    // malloc for VertexOut
                    VertexOut* dev_vertexOut;
                    hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
                    checkCUDAError("Malloc VertexOut Buffer");

                    // ----------Materials-------------

                    // You can only worry about this part once you started to
                    // implement textures for your rasterizer
                    TextureData* dev_diffuseTex = NULL;
                    int diffuseTexWidth = 0;
                    int diffuseTexHeight = 0;
                    if (!primitive.material.empty()) {
                        const tinygltf::Material &mat = scene.materials.at(primitive.material);
                        printf("material.name = %s\n", mat.name.c_str());

                        if (mat.values.find("diffuse") != mat.values.end()) {
                            std::string diffuseTexName = mat.values.at("diffuse").string_value;
                            if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
                                const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
                                if (scene.images.find(tex.source) != scene.images.end()) {
                                    const tinygltf::Image &image = scene.images.at(tex.source);

                                    size_t s = image.image.size() * sizeof(TextureData);
                                    hipMalloc(&dev_diffuseTex, s);
                                    hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);

                                    diffuseTexWidth = image.width;
                                    diffuseTexHeight = image.height;

                                    checkCUDAError("Set Texture Image data");
                                }
                            }
                        }

                        // TODO: write your code for other materails
                        // You may have to take a look at tinygltfloader
                        // You can also use the above code loading diffuse material as a start point
                    }


                    // ---------Node hierarchy transform--------
                    hipDeviceSynchronize();

                    dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    _nodeMatrixTransform<<<numBlocksNodeTransform, numThreadsPerBlock>>>(
                            numVertices,
                            dev_position,
                            dev_normal,
                            matrix,
                            matrixNormal);

                    checkCUDAError("Node hierarchy transformation");

                    // at the end of the for loop of primitive
                    // push dev pointers to map
                    primitiveVector.push_back(PrimitiveDevBufPointers{
                            primitive.mode,
                            primitiveType,
                            numPrimitives,
                            numIndices,
                            numVertices,

                            dev_indices,
                            dev_position,
                            dev_normal,
                            dev_texcoord0,

                            dev_diffuseTex,
                            diffuseTexWidth,
                            diffuseTexHeight,

                            dev_vertexOut	//VertexOut
                    });

                    totalNumPrimitives += numPrimitives;

                } // for each primitive

            } // for each mesh

        } // for each node

    }


    // 3. Malloc for dev_primitives
    {
        sceneInfo.numPrimitives = totalNumPrimitives;
        hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
    }


    // Finally, hipFree raw dev_bufferViews
    {

        std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
        std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());

        //bufferViewDevPointers

        for (; it != itEnd; it++) {
            hipFree(it->second);
        }

        checkCUDAError("Free BufferView Device Mem");
    }

}
