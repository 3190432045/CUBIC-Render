#include "hip/hip_runtime.h"
/**
 * @file      render.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */
#include <map>
#include <string>
#include <vector>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include "util/checkCUDAError.h"
#include "external/include/glm/gtc/quaternion.hpp"
#include "external/include/glm/gtx/transform.hpp"

#include "dataType.h"
#include "renderTool.h"
#include "renderFunc.h"
#include "render.h"


////////////////////////////////////////////////////////////////
///                      Render Pipeline                     ///
////////////////////////////////////////////////////////////////

void Render::render(uchar4 *pbo, const glm::mat4 & M, const glm::mat4 & V, const glm::mat4 & P) {
    dim3 blockSize2d(tileSize, tileSize);
    dim3 blockCount2d((width - 1) / tileSize + 1,(height - 1) / tileSize + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
        int curPrimitiveBeginId = 0; // change static to non-static
		dim3 numThreadsPerBlock(defaultThreadPerBlock);

		auto it = sceneInfo.mesh2PrimitivesMap.begin();
		auto itEnd = sceneInfo.mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

                _vertexTransform<<<numBlocksForVertices, numThreadsPerBlock>>>
                        (p->numVertices,
                         *p,
                         M, V, P,
                         width,
                         height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly<<<numBlocksForIndices, numThreadsPerBlock>>>
					(p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}

    hipMemset(dev_tileBuffer, 0, ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
    checkCUDAError("_clearBuffer");

    // TODO: render
    {
        dim3 numThreadsPerBlock = defaultThreadPerBlock;
        dim3 numBlocks = (sceneInfo.numPrimitives + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x;
        _generateTileBuffer<<<numBlocks,numThreadsPerBlock>>>
                (sceneInfo.numPrimitives,
                 dev_primitives,
                 dev_tileBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("_generateTileBuffer");

        _rasterize<<<blockCount2d,blockSize2d,tileSize*tileSize*sizeof(Fragment)>>>
                (dev_primitives,
                 dev_tileBuffer,
                 dev_fragmentBuffer,
                 width,
                 height,
                 tileSize);
        checkCUDAError("_rasterize");

    }


    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

    // Copy depthbuffer colors into framebuffer
	_fragmentShading<<<blockCount2d, blockSize2d>>>(dev_framebuffer, dev_fragmentBuffer, overrideMaterial, width, height);
	checkCUDAError("fragment shader");

    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    _copyImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}



////////////////////////////////////////////////////////////////
/// Functions that only be called when program start or exit ///
////////////////////////////////////////////////////////////////


/**
 * Called once at the end of the program to free CUDA memory.
 */
void Render::free() {

    // deconstruct primitives attribute/indices device buffer

    auto it(sceneInfo.mesh2PrimitivesMap.begin());
    auto itEnd(sceneInfo.mesh2PrimitivesMap.end());
    for (; it != itEnd; ++it) {
        for (auto p = it->second.begin(); p != it->second.end(); ++p) {
            hipFree(p->dev_indices);
            hipFree(p->dev_position);
            hipFree(p->dev_normal);
            hipFree(p->dev_uv);
            for(int i=0; i<maxTexNum; i++)
            {
                hipFree(p->dev_tex[i].data);
            }

            hipFree(p->dev_verticesOut);


            //TODO: release other attributes and materials
        }
    }

    ////////////

    hipFree(dev_primitives);
    dev_primitives = nullptr;

    hipFree(dev_fragmentBuffer);
    dev_fragmentBuffer = nullptr;

    hipFree(dev_tileBuffer);
    dev_tileBuffer = nullptr;

    hipFree(dev_framebuffer);
    dev_framebuffer = nullptr;

    checkCUDAError("render Free");
}

void Render::init(const tinygltf::Scene & scene,const int &w,const int &h) {

    width = w;
    height = h;
    hipFree(dev_fragmentBuffer);
    hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
    hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_tileBuffer);
    hipMalloc(&dev_tileBuffer, ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
    hipMemset(dev_tileBuffer, 0, ((width + tileSize - 1) / tileSize) * ((height + tileSize - 1) / tileSize) * sizeof(Tile));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("init");

    int totalNumPrimitives = 0; // change static to non-static

    std::map<std::string, BufferByte*> bufferViewDevPointers;

    // 1. copy all `bufferViews` to device memory
    {
        auto it(scene.bufferViews.begin());
        auto itEnd(scene.bufferViews.end());

        for (; it != itEnd; it++) {
            const std::string key = it->first;
            const tinygltf::BufferView &bufferView = it->second;
            if (bufferView.target == 0) {
                continue; // Unsupported bufferView.
            }

            const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

            BufferByte* dev_bufferView;
            hipMalloc(&dev_bufferView, bufferView.byteLength);
            hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

            checkCUDAError("Set BufferView Device Mem");

            bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

        }
    }



    // 2. for each mesh:
    //		for each primitive:
    //			build device buffer of indices, materail, and each attributes
    //			and store these pointers in a map
    {
        std::map<std::string, glm::mat4> nodeString2Matrix;
        auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

        {
            auto it = rootNodeNamesList.begin();
            auto itEnd = rootNodeNamesList.end();
            for (; it != itEnd; ++it) {
                _traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
            }
        }


        // parse through node to access mesh

        auto itNode = nodeString2Matrix.begin();
        auto itEndNode = nodeString2Matrix.end();
        for (; itNode != itEndNode; ++itNode) {

            const tinygltf::Node & N = scene.nodes.at(itNode->first);
            const glm::mat4 & matrix = itNode->second;
            const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

            auto itMeshName = N.meshes.begin();
            auto itEndMeshName = N.meshes.end();

            for (; itMeshName != itEndMeshName; ++itMeshName) {

                const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

                auto res = sceneInfo.mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveBuffer>>(mesh.name, std::vector<PrimitiveBuffer>()));
                std::vector<PrimitiveBuffer> & primitiveVector = (res.first)->second;

                // for each primitive
                for (size_t i = 0; i < mesh.primitives.size(); i++) {
                    const tinygltf::Primitive &primitive = mesh.primitives[i];

                    if (primitive.indices.empty())
                        return;

                    // TODO: add new attributes for your PrimitiveBuffer when you add new attributes
                    VertexIndex* dev_indices = nullptr;
                    glm::vec3* dev_position = nullptr;
                    glm::vec3* dev_normal = nullptr;
                    glm::vec2* dev_texcoord0 = nullptr;

                    // ----------Indices-------------

                    const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
                    const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
                    BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

                    // assume type is SCALAR for indices
                    int n = 1;
                    int numIndices = indexAccessor.count;
                    int componentTypeByteSize = sizeof(VertexIndex);
                    int byteLength = numIndices * n * componentTypeByteSize;

                    dim3 numThreadsPerBlock(defaultThreadPerBlock);
                    dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    hipMalloc(&dev_indices, byteLength);
                    _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                            numIndices,
                            (BufferByte*)dev_indices,
                            dev_bufferView,
                            n,
                            indexAccessor.byteStride,
                            indexAccessor.byteOffset,
                            componentTypeByteSize);


                    checkCUDAError("Set Index Buffer");


                    // ---------Primitive Info-------

                    // Warning: LINE_STRIP is not supported in tinygltfloader
                    int numPrimitives;
                    PrimitiveType primitiveType;
                    switch (primitive.mode) {
                        case TINYGLTF_MODE_TRIANGLES:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices / 3;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_STRIP:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_TRIANGLE_FAN:
                            primitiveType = PrimitiveType::Triangle;
                            numPrimitives = numIndices - 2;
                            break;
                        case TINYGLTF_MODE_LINE:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices / 2;
                            break;
                        case TINYGLTF_MODE_LINE_LOOP:
                            primitiveType = PrimitiveType::Line;
                            numPrimitives = numIndices + 1;
                            break;
                        case TINYGLTF_MODE_POINTS:
                            primitiveType = PrimitiveType::Point;
                            numPrimitives = numIndices;
                            break;
                        default:
                            // output error
                            break;
                    }


                    // ----------Attributes-------------

                    auto it(primitive.attributes.begin());
                    auto itEnd(primitive.attributes.end());

                    int numVertices = 0;
                    // for each attribute
                    for (; it != itEnd; it++) {
                        const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
                        const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

                        int n = 1;
                        if (accessor.type == TINYGLTF_TYPE_SCALAR) {
                            n = 1;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC2) {
                            n = 2;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC3) {
                            n = 3;
                        }
                        else if (accessor.type == TINYGLTF_TYPE_VEC4) {
                            n = 4;
                        }

                        BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
                        BufferByte ** dev_attribute = nullptr;

                        numVertices = accessor.count;
                        int componentTypeByteSize;

                        // Note: since the type of our attribute array (dev_position) is static (float32)
                        // We assume the glTF model attribute type are 5126(FLOAT) here

                        if (it->first.compare("POSITION") == 0) {
                            componentTypeByteSize = sizeof(glm::vec3) / n;
                            dev_attribute = (BufferByte**)&dev_position;
                        }
                        else if (it->first.compare("NORMAL") == 0) {
                            componentTypeByteSize = sizeof(glm::vec3) / n;
                            dev_attribute = (BufferByte**)&dev_normal;
                        }
                        else if (it->first.compare("TEXCOORD_0") == 0) {
                            componentTypeByteSize = sizeof(glm::vec2) / n;
                            dev_attribute = (BufferByte**)&dev_texcoord0;
                        }

                        std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

                        dim3 numThreadsPerBlock(defaultThreadPerBlock);
                        dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                        int byteLength = numVertices * n * componentTypeByteSize;
                        hipMalloc(dev_attribute, byteLength);

                        _deviceBufferCopy<<<numBlocks, numThreadsPerBlock>>>(
                                n * numVertices,
                                *dev_attribute,
                                dev_bufferView,
                                n,
                                accessor.byteStride,
                                accessor.byteOffset,
                                componentTypeByteSize);

                        std::string msg = "Set Attribute Buffer: " + it->first;
                        checkCUDAError(msg.c_str());
                    }

                    // malloc for VertexOut
                    VertexOut* dev_vertexOut;
                    hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
                    checkCUDAError("Malloc VertexOut Buffer");

                    // ----------Materials-------------

                    // You can only worry about this part once you started to
                    // implement textures for your rasterizer
                    MaterialType materialType = Invalid;

                    Tex diffuseTex{nullptr,0,0};
                    Tex specularTex{nullptr,0,0};
                    Tex normalTex{nullptr,0,0};
                    Tex roughnessTex{nullptr,0,0};
                    Tex emissionTex{nullptr,0,0};


                    if (!primitive.material.empty()) {
                        const tinygltf::Material &mat = scene.materials.at(primitive.material);
                        printf("material.name = %s\n", mat.name.c_str());

                        _initTex(scene, mat, "diffuse", diffuseTex);
                        _initTex(scene, mat, "specular", specularTex);
                        _initTex(scene, mat, "normal", normalTex);
                        _initTex(scene, mat, "roughness", roughnessTex);
                        _initTex(scene, mat, "emission", emissionTex);
                    }

                    // Generate material info according to texture;
                    if (diffuseTex.data && specularTex.data && normalTex.data && roughnessTex.data)
                        materialType = PBR;
                    else if (diffuseTex.data)
                        materialType = Tex0;
                    else
                        materialType = Mesh;


                    // ---------Node hierarchy transform--------
                    hipDeviceSynchronize();

                    dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
                    _nodeMatrixTransform<<<numBlocksNodeTransform, numThreadsPerBlock>>>(
                            numVertices,
                            dev_position,
                            dev_normal,
                            matrix,
                            matrixNormal);

                    checkCUDAError("Node hierarchy transformation");

                    // at the end of the for loop of primitive
                    // push dev pointers to map
                    primitiveVector.push_back(PrimitiveBuffer{
                            primitive.mode,
                            primitiveType,
                            materialType,
                            numPrimitives,
                            numIndices,
                            numVertices,

                            dev_indices,
                            dev_position,
                            dev_normal,

                            dev_texcoord0,
                            {diffuseTex,
                             specularTex,
                             normalTex,
                             roughnessTex,
                             emissionTex},

                            dev_vertexOut	//VertexOut
                    });

                    totalNumPrimitives += numPrimitives;

                } // for each primitive

            } // for each mesh

        } // for each node

    }


    // 3. Malloc for dev_primitives
    {
        sceneInfo.numPrimitives = totalNumPrimitives;
        hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
    }


    // Finally, hipFree raw dev_bufferViews
    {

        std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
        std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());

        //bufferViewDevPointers

        for (; it != itEnd; it++) {
            hipFree(it->second);
        }

        checkCUDAError("Free BufferView Device Mem");
    }

}

